#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
//#include <cctype.h>
#include "SyncedMemory.h"

const int difference = ((int) 'a') - ((int) 'A');

#define CHECK {\
	auto e = hipDeviceSynchronize();\
	if (e != hipSuccess) {\
		printf("At " __FILE__ ":%d, %s\n", __LINE__, hipGetErrorString(e));\
		abort();\
	}\
}

__global__ void SomeTransform(char *input_gpu, int fsize) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	/* transfer the first (nBlock*nThread) characters to uppercase if they are lower case */
	if (idx < fsize and input_gpu[idx] != '\n') {
		if (input_gpu[idx] >= 'a' && input_gpu[idx] <= 'z')
			input_gpu[idx] -= difference;
	}
}

int main(int argc, char **argv)
{
	// init, and check
	if (argc != 2) {
		printf("Usage %s <input text file>\n", argv[0]);
		abort();
	}
	FILE *fp = fopen(argv[1], "r");
	if (not fp) {
		printf("Cannot open %s", argv[1]);
		abort();
	}
	// get file size
	fseek(fp, 0, SEEK_END);
	size_t fsize = ftell(fp);
	fseek(fp, 0, SEEK_SET);

	// read files
	MemoryBuffer<char> text(fsize+1);
	auto text_smem = text.CreateSync(fsize);
	CHECK;
	fread(text_smem.get_cpu_wo(), 1, fsize, fp);
	text_smem.get_cpu_wo()[fsize] = '\0';
	fclose(fp);

	// TODO: do your transform here
	char *input_gpu = text_smem.get_gpu_rw();
	// An example: transform the first 64 characters to '!'
	// Don't transform over the tail
	// And don't transform the line breaks
	SomeTransform<<<150, 32>>>(input_gpu, fsize);

	puts(text_smem.get_cpu_ro());
	return 0;
}
